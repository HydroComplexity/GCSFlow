// Copyright (C) 2014, HydroComplexity Group
// All rights reserved.
//
// GPU-BASED CONJUNCTIVE SURFACE-SUBSURFACE FLOW MODEL (GCSFlow)
// GCSFlow model is free software; you can redistribute it and/or modify
// it under the terms of the GNU Lesser General Public License as published
// by the Free Software Foundation; either version 2.1 of the License, or
// (at your option) any later version.
//
// GCSFlow is distributed in the hope that it will be useful, but
// WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY
// or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License
// for more details.
//
// You should have received a copy of the GNU Lesser General Public License
// along with GCSFlow; if not, see <http://www.gnu.org/licenses/>.
//
// Author: levuvietphong@gmail.com (Phong Le)


// This file provides functions to initialize and free all memory in device for 
// the  simulations.

#include <string.h>
#include <stdio.h>
#include <cmath>
#include <vector>

#include "../include/globoverland.h"
#include "../include/globsubsurface.h"
#include "../include/globprocess.h"
#include "../include/ressave.h"
#include "../include/class.h"
#include "../include/variableclass.h"
#include "../include/constantextern.h"


// --------------------------------------------------------------------
// cudaInitializeData()
//    Initialize main variables in main CUDA memory for both overland flow and
//    subsurface flow models.
// --------------------------------------------------------------------
void cudaInitializeData(int M, int N, int P, int time_data, int time_steps) {
  double SIZE3D = M*N*P*sizeof(double);
  double SIZE2D = M*N*sizeof(double);
  // Subsurface model variables
  hipMalloc((void**)&Psi_in_d, SIZE3D);
  hipMalloc((void**)&theta_in_d, SIZE3D);
  hipMalloc((void**)&K_in_d, SIZE3D);
  hipMalloc((void**)&C_in_d, SIZE3D);
  hipMalloc((void**)&Psi_out_d, SIZE3D);
  hipMalloc((void**)&theta_out_d, SIZE3D);
  hipMalloc((void**)&K_out_d, SIZE3D);
  hipMalloc((void**)&Ksat_d, SIZE3D);

  hipMalloc((void**)&eRF, SIZE2D);
  hipMalloc((void**)&IN, SIZE2D);
  hipMalloc((void**)&Psidiff, SIZE2D);
  hipMalloc((void**)&hbottom, SIZE2D);
  hipMalloc((void**)&iter_z_d, M*N*sizeof(int));
  hipMalloc((void**)&iter_x_d, M*P*sizeof(int));
  hipMalloc((void**)&iter_y_d, N*P*sizeof(int));

  // Overland - Surface model variables
  hipMalloc((void**)&Hs_in_d, SIZE2D);
  hipMalloc((void**)&h_d, SIZE2D);
  hipMalloc((void**)&Hs_out_d, SIZE2D);
  hipMalloc((void**)&Ztopo_d, SIZE2D);
  hipMalloc((void**)&mann_d, SIZE2D);
  hipMalloc((void**)&K2w_d, SIZE2D);
  hipMalloc((void**)&K2e_d, SIZE2D);
  hipMalloc((void**)&K2n_d, SIZE2D);
  hipMalloc((void**)&K2s_d, SIZE2D);
  hipMalloc((void**)&u2_d, SIZE2D);
  hipMalloc((void**)&v2_d, SIZE2D);
  
  hipMalloc((void**)&PPT_d, time_steps*sizeof(double));
  hipMalloc((void**)&ET_d, time_steps*sizeof(double));
  hipMalloc((void**)&hout_d, M*time_data*sizeof(double));
  hipMalloc((void**)&vout_d, M*time_data*sizeof(double));
  hipMalloc((void**)&uout_d, M*time_data*sizeof(double));
  hipMalloc((void**)&qss_d, time_steps*sizeof(double));

  // Set variables to 0 for initialization
  hipMemset(eRF, 0, SIZE2D);
  hipMemset(IN, 0, SIZE2D);
  hipMemset(K2w_d, 0, SIZE2D);
  hipMemset(K2e_d, 0, SIZE2D);
  hipMemset(K2n_d, 0, SIZE2D);
  hipMemset(K2s_d, 0, SIZE2D);
  hipMemset(u2_d, 0, SIZE2D);
  hipMemset(v2_d, 0, SIZE2D);

  hipMemset(iter_z_d, 0, M*N*sizeof(int));
  hipMemset(iter_x_d, 0, M*P*sizeof(int));
  hipMemset(iter_y_d, 0, N*P*sizeof(int));
}


// --------------------------------------------------------------------
// InitializeVariables()
//    Initialize variables and initial conditions in overland flow and 
//    subsurface flow models
// --------------------------------------------------------------------
void InitializeVariables(int M, int N, int P, int BSZ, int TSZ) {
  double SIZE3D = M*N*P*sizeof(double);
  double SIZE2D = M*N*sizeof(double);
  vanGenuchtenIntial<<<TSZ, BSZ>>>(theta_in_d, K_in_d, Ksat_d, Psi_in_d, M*N*P);
  PondHeadInit<<<TSZ, BSZ>>>(h_d, M*N);
  SweHInit<<<TSZ, BSZ>>>(Hs_in_d, Ztopo_d, h_d, M*N);
  hipMemcpy(Psi_out_d, Psi_in_d, SIZE3D, hipMemcpyDeviceToDevice);
  hipMemcpy(K_out_d, K_in_d, SIZE3D, hipMemcpyDeviceToDevice);
  hipMemcpy(theta_out_d, theta_in_d, SIZE3D, hipMemcpyDeviceToDevice);
  hipMemcpy(Hs_out_d, Hs_in_d, SIZE2D, hipMemcpyDeviceToDevice);
}


// --------------------------------------------------------------------
// FreeGPUMemory()
//    Free device memory at the end of the simulation
// --------------------------------------------------------------------
void FreeGPUMemory() {
  hipFree(Hs_in_d); hipFree(h_d); hipFree(Hs_out_d); hipFree(Ztopo_d);
  hipFree(Psi_in_d); hipFree(Psi_out_d); hipFree(K_in_d); hipFree(K_out_d);
  hipFree(C_in_d); hipFree(vout_d); hipFree(uout_d); hipFree(hout_d);
  hipFree(eRF); hipFree(PPT_d); hipFree(ET_d); hipFree(IN);
  hipFree(K2w_d); hipFree(K2e_d); hipFree(K2n_d); hipFree(K2s_d);
  hipFree(u2_d); hipFree(v2_d); hipFree(qss_d); hipFree(hbottom);
}


// -----------------------------------------------------------------------------
// :::::::::::::::::::::::::::::::: END OF FILE ::::::::::::::::::::::::::::::::
// -----------------------------------------------------------------------------